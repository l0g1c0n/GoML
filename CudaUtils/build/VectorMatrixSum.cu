
#include <hip/hip_runtime.h>
extern "C" {

__global__ void VectorMatrixSumKernel(double* out, const double* matrix, const double* vector, int rows, int cols) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < rows && j < cols) {
        int index = i * cols + j;
        out[index] = matrix[index] + vector[j];
    }
}

void VectorMatrixSumWrapper(double* out, const double* matrix, const double* vector, int rows, int cols) {
    double *d_matrix, *d_vector, *d_out;
    size_t matrixSize = rows * cols * sizeof(double);
    size_t vectorSize = cols * sizeof(double);

    hipMalloc(&d_matrix, matrixSize);
    hipMalloc(&d_vector, vectorSize);
    hipMalloc(&d_out, matrixSize);

    hipMemcpy(d_matrix, matrix, matrixSize, hipMemcpyHostToDevice);
    hipMemcpy(d_vector, vector, vectorSize, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((rows + threadsPerBlock.x - 1) / threadsPerBlock.x, (cols + threadsPerBlock.y - 1) / threadsPerBlock.y);

    VectorMatrixSumKernel<<<blocksPerGrid, threadsPerBlock>>>(d_out, d_matrix, d_vector, rows, cols);

    hipMemcpy(out, d_out, matrixSize, hipMemcpyDeviceToHost);

    hipFree(d_matrix);
    hipFree(d_vector);
    hipFree(d_out);
}

}

