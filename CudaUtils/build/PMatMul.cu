
#include <hip/hip_runtime.h>
extern "C" {

__global__ void PMatMulKernel(double* C, const double* A, const double* B, int M, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < M && j < N) {
        int index = i * N + j;
        C[index] = A[index] * B[index];
    }
}

void PMatMulWrapper(double* C, const double* A, const double* B, int M, int N) {
    double *d_A, *d_B, *d_C;
    size_t size = M * N * sizeof(double);

    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((M + threadsPerBlock.x - 1) / threadsPerBlock.x, (N + threadsPerBlock.y - 1) / threadsPerBlock.y);

    PMatMulKernel<<<blocksPerGrid, threadsPerBlock>>>(d_C, d_A, d_B, M, N);

    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

}

