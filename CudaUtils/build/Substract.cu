
#include <hip/hip_runtime.h>
extern "C" {

__global__ void SubtractKernel(double* out, const double* a, const double* b, int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < size) {
        out[i] = a[i] - b[i];
    }
}

void SubtractWrapper(double* out, const double* a, const double* b, int size) {
    double *d_a, *d_b, *d_out;
    size_t dataSize = size * sizeof(double);

    hipMalloc(&d_a, dataSize);
    hipMalloc(&d_b, dataSize);
    hipMalloc(&d_out, dataSize);

    hipMemcpy(d_a, a, dataSize, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, dataSize, hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;

    SubtractKernel<<<blocksPerGrid, threadsPerBlock>>>(d_out, d_a, d_b, size);

    hipMemcpy(out, d_out, dataSize, hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);
}

}

